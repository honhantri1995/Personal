#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <assert.h>

// Cuda
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "benchmark.h"

#define N       1000000000      // 1 billion

__global__ void vector_add(float *out, float *a, float *b, int n)
{
    for(int i = 0; i < n; i ++) {
        out[i] = a[i] + b[i];
    }
}

int main()
{
    Benchmark::startClock();

    float *a, *b, *out;
    float *d_a, *d_b, *d_out; 

    // Allocate host memory
    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    // Initialize host arrays
    for(int i = 0; i < N; i++) {
        a[i] = 1.0f;
        b[i] = 2.0f;
    }

    // Allocate device memory
    hipMalloc((void**)&d_a, sizeof(float) * N);
    hipMalloc((void**)&d_b, sizeof(float) * N);
    hipMalloc((void**)&d_out, sizeof(float) * N);

    // Transfer data from host to device memory
    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

    // Executing kernel
    // Using 1 block, 1 thread
    // vector_add<<<1,1>>>(d_out, d_a, d_b, N);

    // Executing kernel
    // Using multiple blocks, multiple thread
    int block_size = 256;
    int grid_size = ((N + block_size) / block_size);
    vector_add<<<grid_size,block_size>>>(d_out, d_a, d_b, N);

    // FIXME: Why the same performance?

    // Transfer data back to host memory
    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);

    // Print values
    printf("Sum is: %f\n", out[0]);
    printf("Number of retry is: %d\n", N);

    // Deallocate device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    // Deallocate host memory
    free(a); 
    free(b); 
    free(out);
    
    double duration = Benchmark::stopClock();
    Benchmark::print("Addition on GPU", duration, "");
}
