#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <iostream>
using namespace std;

// Cuda
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "benchmark.h"

#define ITEM_NUM       100000000      // 100 million

__global__ void _addVector(uint *result, uint *a, uint *b, int size)
{
    for(int i = 0; i < size; i ++) {
        result[i] = a[i] + b[i];
    }
}

void addVector(uint first, uint second)
{
    uint *a, *b, *result;
    uint *cd_a, *cd_b, *cd_result; 

    // Allocate host memory
    a       = (uint*)malloc(sizeof(uint) * ITEM_NUM);
    b       = (uint*)malloc(sizeof(uint) * ITEM_NUM);
    result  = (uint*)malloc(sizeof(uint) * ITEM_NUM);

    // Initialize host arrays
    for(int i = 0; i < ITEM_NUM; i++) {
        a[i] = first;
        b[i] = second;
    }

    // Allocate device memory
    hipMalloc((void**)&cd_a, sizeof(uint) * ITEM_NUM);
    hipMalloc((void**)&cd_b, sizeof(uint) * ITEM_NUM);
    hipMalloc((void**)&cd_result, sizeof(uint) * ITEM_NUM);

    // Transfer data from host to device memory
    hipMemcpy(cd_a, a, sizeof(uint) * ITEM_NUM, hipMemcpyHostToDevice);
    hipMemcpy(cd_b, b, sizeof(uint) * ITEM_NUM, hipMemcpyHostToDevice);

    // Executing kernel
    // Using 1 block, 1 thread
    // _addVector<<<1,1>>>(cd_result, cd_a, cd_b, ITEM_NUM);

    // Executing kernel
    // Using multiple blocks, multiple thread
    int block_size = 256;
    int grid_size = ((ITEM_NUM + block_size) / block_size);
    _addVector<<<grid_size,block_size>>>(cd_result, cd_a, cd_b, ITEM_NUM);
    hipDeviceSynchronize();

    // Transfer data back to host memory
    hipMemcpy(result, cd_result, sizeof(uint) * ITEM_NUM, hipMemcpyDeviceToHost);

    // Print result
    cout << "Sum is: " << result[0] << endl;

    // Deallocate device memory
    hipFree(cd_a);
    hipFree(cd_b);
    hipFree(cd_result);

    // Deallocate host memory
    free(a); 
    free(b); 
    free(result);
}

int main()
{
    Benchmark benchmark;
    benchmark.startClock();

    addVector(123456789, 987654321);

    benchmark.print("Addition on GPU", benchmark.stopClock(), "");
}
